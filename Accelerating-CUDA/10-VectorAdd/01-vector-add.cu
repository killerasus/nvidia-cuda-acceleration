
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}


void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for(int i = index; i < N; i += stride)
    result[i] = a[i] + b[i];
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  if(checkCuda(hipMallocManaged(&a, size)) != hipSuccess)
    exit(1);
    
  if(checkCuda(hipMallocManaged(&b, size)) != hipSuccess)
  {
    hipFree(a);
    exit(1);
  }
  
  if(checkCuda(hipMallocManaged(&c, size)) != hipSuccess)
  {
    hipFree(a);
    hipFree(b);
    exit(1);
  }

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threads_per_block = 256;

  // Ensure there are at least `N` threads in the grid, but only 1 block's worth extra
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  hipError_t syncErr, asyncErr;
  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);

  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  if(checkCuda(syncErr) != hipSuccess || checkCuda(asyncErr) != hipSuccess)
  {
    hipFree(a);
    hipFree(b);
    hipFree(c);
    exit(1);
  }

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
